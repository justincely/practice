
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 1000000
#define ARRAY_SIZE 100
#define BLOCK_WIDTH 1000

//------------------------------------------------------------------------------

void print_array(int *array, int size) {
  printf("{ ");

  for (int i=0; i<size; i++) {
    printf("%d ", array[i]);
  }

  printf(" }");
}

//------------------------------------------------------------------------------

__global__ void increment_naive(int *g) {
  // determine thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // Wrap thread to array size
  i = i % ARRAY_SIZE;
  g[i] = g[i] + 1;
}

//------------------------------------------------------------------------------

__global__ void increment_atomic(int *g) {
  // determine thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // wrap thread to array size
  i = i % ARRAY_SIZE;
  atomicAdd(&g[i], 1);
}

//------------------------------------------------------------------------------

int main(int argc, char **argv) {
  printf("%d threads in %d blocks writing %d elements\n",
         NUM_THREADS,
         NUM_THREADS / BLOCK_WIDTH,
         ARRAY_SIZE);

  // array on host memory
  int h_array[ARRAY_SIZE];
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

  // array on GPU
  int *d_array;
  hipMalloc((void **) &d_array, ARRAY_BYTES);
  hipMemset((void *) d_array, 0, ARRAY_BYTES);

  //increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
  increment_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);


  // copy results back from GPU
  hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);

  print_array(h_array, ARRAY_SIZE);

  // free GPU memory
  hipFree(d_array);

  return 0;

}
